#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <sys/time.h>

#include "kernel.h"

const int NUM  = 2;
const int SIZE = 1;
#define DATATYPE float
hipStream_t UTIL_GPU_Stream[NUM+1];


DATATYPE *A;

void UTIL_CreateStreams(){
    UTIL_GPU_Stream[0] = 0;
    int i;
    for(i = 1; i <= NUM; i++) {
        hipStreamCreate(&UTIL_GPU_Stream[i]);
    }
}
void UTIL_deleteStreams(){
    int i;
    for(i = 1; i <= NUM; i++) {
        hipStreamDestroy(UTIL_GPU_Stream[i]);
    }
}

void UTIL_MallocMem(){
    hipMalloc(&A, sizeof(DATATYPE)*SIZE);
}
void UTIL_FreeMem(){
    hipFree(A);
}


extern "C" void init(){
    UTIL_CreateStreams();
    UTIL_MallocMem();
}
extern "C" void end(){
    UTIL_deleteStreams();
    UTIL_FreeMem();
}

#define N_ITERATIONS (1024000)
__global__ void FFMA(DATATYPE *dst, DATATYPE half){
    DATATYPE a1 = 1;
    DATATYPE a2 = 2;
    DATATYPE a3 = 3;
    DATATYPE a4 = 4;
    DATATYPE a5 = 5;
    DATATYPE a6 = 6;
    DATATYPE b = half+1;

    #pragma unroll 128
    for( int i = 0; i < N_ITERATIONS; i ++ )
    {
      a1 = a1*b+b;
      a2 = a2*b+b;
      a3 = a3*b+b;
      a4 = a4*b+b;
      a5 = a1*a3+a5;
      a6 = a2*a4+a6;
    }
    dst[threadIdx.x + blockDim.x*blockIdx.x] = a5+a6;
}

extern "C" void UTIL_None(int streamId){
    FFMA<<<1, 1, 0, UTIL_GPU_Stream[streamId]>>>(A, 0.5);
}

extern "C" int getUtilization()
{
    FILE * fp;
    char buffer[100];
    fp = popen("nvidia-smi | grep \"%\" | awk '{print $13}'","r");
    fgets(buffer, sizeof(buffer), fp);
    printf("GPU Utilization: %s", buffer);
    pclose(fp);
    return atoi(buffer);
}
